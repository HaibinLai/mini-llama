#include <stdio.h>
#include <hip/hip_runtime.h>

// CUDA kernel for vector addition
__global__ void vectorAdd(const float *a, const float *b, float *c, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        c[idx] = a[idx] + b[idx];
    }
}

int main() {
    const int N = 1024; // Vector size
    size_t size = N * sizeof(float);

    // Host vectors
    float *h_a = (float *)malloc(size);
    float *h_b = (float *)malloc(size);
    float *h_c = (float *)malloc(size);

    // Initialize input vectors
    for (int i = 0; i < N; i++) {
        h_a[i] = rand() / (float)RAND_MAX;
        h_b[i] = rand() / (float)RAND_MAX;
    }

    // Device vectors
    float *d_a, *d_b, *d_c;
    hipError_t err;
    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);

    // Copy inputs to device
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    // Launch kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, N);
    
    // Copy result back to host
    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);


    // Verify result
    for (int i = 0; i < N; i++) {
        if (fabs(h_a[i] + h_b[i] - h_c[i]) > 1e-5) {
            printf("Verification failed at index %d!\n", i);
            break;
        }
    }

    // Print first few results
    printf("First 5 results of vector addition:\n");
    for (int i = 0; i < 5; i++) {
        printf("C[%d] = %.2f + %.2f = %.2f\n", i, h_a[i], h_b[i], h_c[i]);
    }
    printf("Hello CUDA! Vector addition completed.\n");

    // Cleanup
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(h_a);
    free(h_b);
    free(h_c);

    return 0;
}